#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>

#include "brdf.hpp"
#include "error.hpp"
#include "gpuRender.hpp"
#include "random.hpp"

#define INV2PI (1.0f / (2.0f * M_PI))
#define INVPI (1.0f / M_PI)

__device__ vec2 GPUGeometry::getTexCoord(uint idx, vec2 barycentric) const {
  if (hasTexCoords) {
    vec2 v0 = texCoords[3u * idx];
    vec2 v1 = texCoords[3u * idx + 1u];
    vec2 v2 = texCoords[3u * idx + 2u];
    return barycentric.x * v1 + barycentric.y * v2 + (1.0f - (barycentric.x + barycentric.y)) * v0;
  }

  return vec2{0};
}

__device__ vec3 GPUGeometry::getNormal(uint idx, vec2 barycentric) const {
  if (hasNormals) {
    vec3 v0 = vertexNormals[3u * idx];
    vec3 v1 = vertexNormals[3u * idx + 1u];
    vec3 v2 = vertexNormals[3u * idx + 2u];
    return barycentric.x * v1 + barycentric.y * v2 + (1.0f - (barycentric.x + barycentric.y)) * v0;
  }

  return faceNormals[idx];
}

// Find the distance to the closest intersection, the index of the primitive and the number of BVH tests.
__device__ void GPUGeometry::intersect(Ray& ray, HitRecord& hitRecord, uint& count) const {
  intersectBVH(ray, bvh, primitives, indices, 0, hitRecord, count);
}

__device__ void GPUMesh::intersect(Ray& ray, HitRecord& hitRecord, uint& count) const {
  Ray transformedRay = ray;
  transformedRay.origin = invModelMatrix * vec4(ray.origin, 1.0f);
  // Not normalized to handle scale transform
  transformedRay.direction = invModelMatrix * vec4(ray.direction, 0.0f);
  transformedRay.invDirection = 1.0f / transformedRay.direction;

  geometry->intersect(transformedRay, hitRecord, count);
  ray.t = transformedRay.t;
}

//-------------------------------- Rotations --------------------------------

__device__ inline vec3 rotate(vec3 p, vec4 q) {
  return 2.0f * cross(vec3(q), p * q.w + cross(vec3(q), p)) + p;
}
__device__ inline vec3 rotateX(vec3 p, float angle) {
  return rotate(p, vec4(sin(angle / 2.0), 0.0, 0.0, cos(angle / 2.0)));
}
__device__ inline vec3 rotateY(vec3 p, float angle) {
  return rotate(p, vec4(0.0, sin(angle / 2.0), 0.0, cos(angle / 2.0)));
}
__device__ inline vec3 rotateZ(vec3 p, float angle) {
  return rotate(p, vec4(0.0, 0.0, sin(angle / 2.0), cos(angle / 2.0)));
}

__device__ vec3 getEnvironment(const GPUImage* environment, const vec3& direction) {
  // Rotate environment map
  vec3 sampleDir = normalize(rotateY(direction, -M_PI));
  uint u = environment->width * (atan2f(sampleDir.z, sampleDir.x) * INV2PI + 0.5f);
  uint v = environment->height * acosf(sampleDir.y) * INVPI;
  uint idx = min(u + v * environment->width, (environment->width * environment->height) - 1);

  return 0.5f * (*environment)[idx];
}

__device__ vec3 getIllumination(Ray& ray, const GPUScene* scene, const GPUImage* environment,
                                uint& rngState, int& bounceCount, uint& testCount) {
  // Initialize light to white and track attenuation
  vec3 col{1};

  for (uint i = 0; i < bounceCount; i++) {
    HitRecord closestHit{};
    uint meshIdx = scene->intersect(ray, closestHit, testCount);

    if (ray.t < FLT_MAX) {
      const GPUMesh& mesh{scene->meshes[meshIdx]};

      vec3 p = ray.origin + ray.direction * ray.t;
      vec3 N = normalize(vec3(mesh.normalMatrix * vec4(mesh.geometry->getNormal(closestHit.hitIndex, closestHit.barycentric), 0.0f)));
      if (dot(ray.direction, N) > 0.0f) {
        N = -N;
      }

      const float metalness{mesh.material->metalness};
      const float roughness{mesh.material->roughness};
      vec2 uv = mesh.geometry->getTexCoord(closestHit.hitIndex, closestHit.barycentric);
      const vec3 albedo = mesh.material->getAlbedo(uv);
      const vec3 emissive = mesh.material->getEmissive(uv);

      vec3 F0 = mix(mesh.material->F0, albedo, metalness);

      vec3 sampleDir{};
      vec3 localCol{};

      //--------------------- Specular ------------------------

      vec2 Xi = getRandomVec2(rngState);
      // Get a random halfway vector around the surface normal (in world space)
      vec3 H = importanceSampleGGX(Xi, N, roughness);

      vec3 V = -ray.direction;

      // Generate sample direction as view ray reflected around h (note sign)
      sampleDir = normalize(reflect(-V, H));

      float NdotL = dot_c(N, sampleDir);
      float NdotV = dot_c(N, V);
      float NdotH = dot_c(N, H);
      float VdotH = dot_c(V, H);

      vec3 F = fresnel(VdotH, F0);
      float G = smiths(NdotV, NdotL, roughness);

      /*

          The following can be simplified as the D term and many dot products cancel out

          float D = distribution(NdotH, roughness);

          // Cook-Torrance BRDF
          vec3 brdfS = D * F * G / max(0.0001, (4.0 * NdotV * NdotL));

          float pdfSpecular = (D * NdotH) / (4.0 * VdotH);
          vec3 specular = (L(sampleDir) * brdfS * NdotL) / pdfSpecular;

      */

      // Simplified from the above

      localCol = (F * G * VdotH) / (NdotV * NdotH);

      if (metalness < 1.0f && getRandomFloat(rngState) > F.x) {
        //--------------------- Diffuse ------------------------
        vec2 Xi = getRandomVec2(rngState);
        vec3 kD = (1.0f - F) * (1.0f - metalness);

        sampleDir = mix(sampleDir, importanceSampleCosine(Xi, N), kD.x);

        /*
            The discrete Riemann sum for the lighting equation is
            1/N * Σ(brdf(l, v) * L(l) * dot(l, n)) / pdf(l))
            Lambertian BRDF is c/PI and the pdf for cosine sampling is dot(l, n)/PI
            PI term and dot products cancel out leaving just c * L(l)
        */
        localCol += kD * albedo;
      }

      col *= localCol;
      col += emissive;
      ray = Ray{p + 1e-4f * N, sampleDir, 1.0f / sampleDir, FLT_MAX};
    } else {
      col *= getEnvironment(environment, ray.direction);
      return col;
    }
  }

  return vec3(0);
}

__global__ void printTLAS(GPUScene* scene, uint size) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= 1) {
    return;
  }
  Ray ray{};
  HitRecord closestHit{};
  uint bvhTests{0u};

  scene->intersect(ray, closestHit, bvhTests);
}

__global__ void render(GPUScene* scene, Camera camera, GPUImage* image, GPUImage* environment,
                       uint samples, int maxBounces, bool renderBVH) {
  vec2 fragCoord = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

  if (fragCoord.x >= image->width || fragCoord.y >= image->height) {
    return;
  }

  uint idx = (uint)(fragCoord.y) * image->width + (uint)(fragCoord.x);

  vec2 resolution{image->width, image->height};
  vec3 col{0};
  uint rngState{1023u + idx};

  for (uint s = 0u; s < (renderBVH ? 1u : samples); s++) {
    vec2 fC = fragCoord;
    if (!renderBVH && samples > 1u) {
      // Jitter position for antialiasing
      fC += 0.5f * (2.0f * getRandomVec2(rngState) - 1.0f);
    }

    Ray ray{.origin = camera.position};
    ray.direction = rayDirection(resolution, camera.fieldOfView, fC);
    ray.direction = normalize(viewMatrix(camera.position, camera.target, camera.up) * ray.direction);
    ray.invDirection = 1.0f / ray.direction;
    ray.t = FLT_MAX;

    uint bvhTests = 0u;
    if (renderBVH) {
      HitRecord closestHit{};
      // Get number of BVH tests for primary ray
      scene->intersect(ray, closestHit, bvhTests);
      (*image)[idx] = vec3(bvhTests);
    } else {
      // Path trace scene
      int bounces = maxBounces;
      col += getIllumination(ray, scene, environment, rngState, bounces, bvhTests);
    }
  }

  if (!renderBVH) {
    // Average result
    col /= samples;
    // Tonemapping
    col *= 1.0f - vec3{expf(-col.r), expf(-col.g), expf(-col.b)};
    // Gamma correction
    col = pow(col, vec3{1.0f / 2.2f});
    // Output data
    (*image)[idx] = col;
  }
}

void renderGPU(
    const Scene& scene,
    const std::vector<std::shared_ptr<Geometry>>& geometryPool,
    const std::vector<std::shared_ptr<Material>>& materialPool,
    const Camera& camera,
    Image& image,
    const Image& environment,
    const uint samples,
    const int maxBounces,
    const bool renderBVH) {
  // Make copy of render target image
  GPUImage gpuImage{image};
  GPUImage* imageDevicePtr;
  CHECK_CUDA_ERROR(hipMalloc(&imageDevicePtr, sizeof(GPUImage)));
  CHECK_CUDA_ERROR(hipMemcpy(imageDevicePtr, &gpuImage, sizeof(GPUImage), hipMemcpyHostToDevice));

  // Copy environment
  GPUImage gpuEnvironment{environment};
  GPUImage* environmentDevicePtr;
  CHECK_CUDA_ERROR(hipMalloc(&environmentDevicePtr, sizeof(GPUImage)));
  CHECK_CUDA_ERROR(hipMemcpy(environmentDevicePtr, &gpuEnvironment, sizeof(GPUImage), hipMemcpyHostToDevice));

  // While the CPU code has objects managing their meshes and geometries using STL containers (and automatic RAII), the GPU code
  // should have separate pointer handles for all data to allocate and clean up memory correctly.

  // Geometry and material data is cleaned up when the destructors of their objects are called
  // Cannot use STL smart pointers here but Thrust probably has a better solution to avoid raw pointers
  std::vector<GPUGeometry*> gpuGeometryPool;
  gpuGeometryPool.reserve(geometryPool.size());

  std::vector<GPUMaterial*> gpuMaterialPool;
  gpuMaterialPool.reserve(gpuMaterialPool.size());

  for (const auto& geometryPtr : geometryPool) {
    gpuGeometryPool.emplace_back(new GPUGeometry(*geometryPtr));
  }

  for (const auto& materialPtr : materialPool) {
    gpuMaterialPool.emplace_back(new GPUMaterial(*materialPtr));
  }

  // Containers for device pointers handed to GPUMesh objects. These do not manage data
  std::vector<GPUGeometry*> geometryDevicePtr;
  geometryDevicePtr.reserve(geometryPool.size());

  std::vector<GPUMaterial*> materialDevicePtr;
  materialDevicePtr.reserve(gpuMaterialPool.size());

  for (const auto& geometry : gpuGeometryPool) {
    GPUGeometry* gpuGeometryPtr;
    CHECK_CUDA_ERROR(hipMalloc(&gpuGeometryPtr, sizeof(GPUGeometry)));
    CHECK_CUDA_ERROR(hipMemcpy(gpuGeometryPtr, geometry, sizeof(GPUGeometry), hipMemcpyHostToDevice));
    geometryDevicePtr.emplace_back(gpuGeometryPtr);
  }

  for (const auto& material : gpuMaterialPool) {
    GPUMaterial* gpuMaterialPtr;
    CHECK_CUDA_ERROR(hipMalloc(&gpuMaterialPtr, sizeof(GPUMaterial)));
    CHECK_CUDA_ERROR(hipMemcpy(gpuMaterialPtr, material, sizeof(GPUMaterial), hipMemcpyHostToDevice));
    materialDevicePtr.emplace_back(gpuMaterialPtr);
  }

  // The order of geometries and materials in the CPU and GPU pools is the same. Find the matching pointers and connect GPU-side data
  /*
    Create pointers for each GPUGeometry and GPUMaterial (the data already exists on the GPU)
    Find the index match in the two CPU containers
    Assign correct pointers to GPU meshes on creation
  */
  std::vector<GPUMesh> gpuMeshes;
  gpuMeshes.reserve(scene.meshes.size());

  for (const auto& mesh : scene.meshes) {
    uint geometryIdx = std::distance(geometryPool.begin(), std::find(geometryPool.begin(), geometryPool.end(), mesh.geometry));
    uint materialIdx = std::distance(materialPool.begin(), std::find(materialPool.begin(), materialPool.end(), mesh.material));

    gpuMeshes.push_back(GPUMesh(mesh, geometryDevicePtr[geometryIdx], materialDevicePtr[materialIdx]));
  }

  // Copy scene
  GPUScene gpuScene{scene, gpuMeshes};
  GPUScene* sceneDevicePtr;
  CHECK_CUDA_ERROR(hipMalloc(&sceneDevicePtr, sizeof(GPUScene)));
  CHECK_CUDA_ERROR(hipMemcpy(sceneDevicePtr, &gpuScene, sizeof(GPUScene), hipMemcpyHostToDevice));

  // Determine number of threads and blocks covering all pixels
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(ceil((float)(gpuImage.width) / (float)(threadsPerBlock.x)),
                 ceil((float)(gpuImage.height) / (float)(threadsPerBlock.y)));

  // printTLAS<<<dim3(10), dim3(1)>>>(sceneDevicePtr, 10);
  // hipDeviceSetLimit(hipLimitStackSize, 1e4);

  /* Timer */ auto start = std::chrono::steady_clock::now();

  /* Call Kernel */ render<<<numBlocks, threadsPerBlock>>>(sceneDevicePtr, camera, imageDevicePtr, environmentDevicePtr, samples, maxBounces, renderBVH);

  CHECK_LAST_CUDA_ERROR();

  hipDeviceSynchronize();

  /* Timer */ std::chrono::duration<double> elapsed_seconds = std::chrono::steady_clock::now() - start;
  /* Timer */ std::cout << "\nRender time: " << std::floor(elapsed_seconds.count() * 1e4f) / 1e4f << " s\n";

  // Copy data back to host
  CHECK_CUDA_ERROR(hipMemcpy(image.data.data(), gpuImage.data, image.data.size() * sizeof(vec3), hipMemcpyDeviceToHost));

  // Free device pointers
  CHECK_CUDA_ERROR(hipFree(imageDevicePtr));
  CHECK_CUDA_ERROR(hipFree(environmentDevicePtr));
  CHECK_CUDA_ERROR(hipFree(sceneDevicePtr));

  for (const auto& geometryPtr : geometryDevicePtr) {
    CHECK_CUDA_ERROR(hipFree(geometryPtr));
  }

  for (const auto& materialPtr : materialDevicePtr) {
    CHECK_CUDA_ERROR(hipFree(materialPtr));
  }

  for (const auto& geometry : gpuGeometryPool) {
    delete geometry;
  }

  for (const auto& material : gpuMaterialPool) {
    delete material;
  }
}