#include "hip/hip_runtime.h"
#include "camera.hpp"

// Generate default ray for a fragment based on its position, the image and the camera
__device__ __host__ vec3 rayDirection(const vec2& resolution, float fieldOfView, const vec2& fragCoord) {
  vec2 xy = fragCoord - 0.5f * resolution;
  float z = (0.5f * resolution.y) / tan(0.5f * radians(fieldOfView));
  return normalize(vec3(xy, -z));
}

__device__ __host__ mat3 viewMatrix(vec3 camera, vec3 at, vec3 up) {
  vec3 zaxis = normalize(at - camera);
  vec3 xaxis = normalize(cross(zaxis, up));
  vec3 yaxis = cross(xaxis, zaxis);

  return mat3(xaxis, yaxis, -zaxis);
}